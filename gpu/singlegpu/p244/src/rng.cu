#include "hip/hip_runtime.h"
#include "api.h"

__global__ void setup_rand(hiprandStatePhilox4_32_10_t *state, uint64_t seed) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(seed, i, 0, &state[i]);
}

__device__ void init_rand(hiprandStatePhilox4_32_10_t *state, uint64_t seed, int seq) {
    int t = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    if (seq)
        hiprand_init(seed, seq, 0, &state[t]);
    else
        hiprand_init(seed, t, 0, &state[t]);
}

/* IMPORTANT: this function just generates random numbers for 2, 3, and 4 words */
__device__ void randombytes(fp_t a, uint32_t mask, uint32_t nbits, hiprandStatePhilox4_32_10_t *state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    uint4 r[2];

    /* Copy state to local memory for efficiency */
    hiprandStatePhilox4_32_10_t localState = state[i];

    /* Generate pseudo-random unsigned ints */
    r[0] = curand4(&localState);
    r[1] = curand4(&localState);
    
    /* Store results */
    if (nbits <= 32) {
        a[0] = (uint64_t)r[0].x & mask;
    } else if (nbits <= 64) {
        a[0] = (uint64_t)r[0].x;
        a[0] ^= ((uint64_t)r[0].y & mask) << 32;
    } else if(nbits <= 96) {
        a[0] = (uint64_t)r[0].x;
        a[0] ^= ((uint64_t)r[0].y) << 32;
        a[1] = r[0].z & mask;
    } else if (nbits <= 128) {
        a[0] = (uint64_t)r[0].x;
        a[0] ^= ((uint64_t)r[0].y) << 32;
        a[1] = (uint64_t)r[0].z;
        a[1] ^= ((uint64_t)r[0].w & mask) << 32;
    }

    else if (nbits <= 160) {
        a[0] = (uint64_t)r[0].x;
        a[0] ^= ((uint64_t)r[0].y) << 32;
        a[1] = (uint64_t)r[0].z;
        a[1] ^= (uint64_t)r[0].w << 32;
        a[2] = (uint64_t)r[1].x & mask;
    } else if (nbits <= 192) {
        a[0] = (uint64_t)r[0].x;
        a[0] ^= ((uint64_t)r[0].y) << 32;
        a[1] = (uint64_t)r[0].z;
        a[1] ^= (uint64_t)r[0].w << 32;
        a[2] = (uint64_t)r[1].x;
        a[2] ^= ((uint64_t)r[1].y & mask) << 32;
    }

    else {
        a[0] = (uint64_t)r[0].x;
        a[0] ^= ((uint64_t)r[0].y) << 32;
        a[1] = (uint64_t)r[0].z;
        a[1] ^= (uint64_t)r[0].w << 32;
        a[2] = (uint64_t)r[1].x;
        a[2] ^= (uint64_t)r[1].y << 32;
        a[3] = (uint64_t)r[1].z;
        a[3] ^= ((uint64_t)r[1].w & mask) << 32;
    }


    /* Copy state back to global memory */
    state[i] = localState;
}


